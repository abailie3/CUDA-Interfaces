#include "hip/hip_runtime.h"
/*
======== CUDA matrix operations v0.2 ========
			By: Austin Bailie

Various matrix operations computed on the GPU.
Over-commented for educational purposes

Adapted from:
	-nVidia's CUDA Programming guide
	-Other credits appear in their respective spots
===========================================
*/
/*
============ Change Log ===================
v0: 1/15/2017		- original

v0.01: 1/15/2017	- added transpose

v0.1: 1/21/2016		- added various matrix math functions
					- added neural network architecture:
						- added logistic2D kernel
						- added lmatSend2D
						- added nodeRetrieve
						- added processNodes
						- added layersetup
						- added hiddenSetup
					- current neural network support runs with 0 errors on Cuda-memcheck

v0.2: 1/29/2017		- implemented working neural network functionality:
						- added nodeBackwardLog kernel
						- added outPivotLog kernel
						- added updateNodes kernel
						- added sendActual support function
						- addded uNodes support function
						- added changeIn support function
						- added pNodes process function
						- tweaked most of the previously added neural network architecture
						- changed main function to run the neural network
					- current technology will successfuly perform batch gradient descent
					  with 0 errors on Cuda-memcheck

===========================================
*/
//required headers:
#include "nodeSet.h" //v0; includes stdio.h and now math.h
//optional headers:
#include <stdlib.h> //so malloc doesn't show as error... will compile without

#define BLKSZ 16

__global__ void mAddKernel2D(Mat2D d_a, Mat2D d_b, Mat2D d_o) {
	/*CUDA kernel for 2d matrix addition
	  needs: nodeSet.h,
	  I don't really need hip/hip_runtime.h or device_launch parameters, but they prevent errors... will compile without
	*/
	int r = blockDim.y * blockIdx.y + threadIdx.y; //getting row based on CUDA thread/block index
	int c = blockDim.x * blockIdx.x + threadIdx.x; //getting column based on CUDA thread/block index
	
	//if out-of-bounds stop
	if (r > d_o.rows || c > d_o.columns) return;

	//add
	d_o.cells[r * d_o.columns + c] = d_a.cells[r * d_o.columns + c] + d_b.cells[r * d_o.columns + c];
	//~~ALB
}

__global__ void mSubKernel2D(Mat2D d_a, Mat2D d_b, Mat2D d_o) {
	/*CUDA kernel for 2d matrix subtraction
	  needs: nodeSet.h,
	  I don't really need hip/hip_runtime.h or device_launch parameters, but they prevent errors... will compile without
	*/
	int r = blockDim.y * blockIdx.y + threadIdx.y; //getting row based on CUDA thread/block index
	int c = blockDim.x * blockIdx.x + threadIdx.x; //getting column based on CUDA thread/block index

	//if out-of-bounds stop
	if (r > d_o.rows || c > d_o.columns) return;

	//subtract
	d_o.cells[r * d_o.columns + c] = d_a.cells[r * d_o.columns + c] - d_b.cells[r * d_o.columns + c];
}

__global__ void mTransKernel2D(Mat2D d_i, Mat2D d_o) {
	/*CUDA kernel for 2d matrix subtraction
	  needs: nodeSet.h
	  I don't really need hip/hip_runtime.h or device_launch parameters, but they prevent errors... will compile without
	*/
	int r = blockIdx.y * blockDim.y + threadIdx.y; //getting row based on CUDA thread/block index
	int c = blockIdx.x * blockDim.x + threadIdx.x; //getting column based on CUDA thread/block index

	//if out-of-bounds stop
	if (r > d_o.rows || c > d_o.columns) return;
	
	//switch rows and columns
	d_o.cells[r * d_o.columns + c] = d_i.cells[c * d_i.columns + r];
}

__global__ void mMultKernel2D(Mat2D d_a, Mat2D d_b, Mat2D d_c) { 
	/*CUDA kernel for 2D matrix multiplication
	  needs: nodeSet.h,
	  I don't really need hip/hip_runtime.h or device_launch parameters, but they prevent errors... will compile without
	  
	  Adapted from:
	  -Robert Hochberg (1/24/16): http://bit.ly/2iA8jDc
	  */

	float oVal = 0;//output value

	int r = blockIdx.y * blockDim.y + threadIdx.y; //getting row based on CUDA thread/block index
	int c = blockIdx.x * blockDim.x + threadIdx.x; //getting column based on CUDA thread/block index

    //if out-of-bounds stop
	//if (r > d_c.rows || c > d_c.columns) return; So I changed my index out of bounds protection, see below

	//add up each A(r,i)*B(i,c) into oVal
	if (r < d_c.rows && c < d_c.columns) {
		for (int i = 0; i < d_a.columns; ++i)
			oVal += d_a.cells[r * d_a.columns + i] * d_b.cells[i*d_b.columns + c];
	}
	else {
		return;
	}
	//assign the oVal to the output matrix
	d_c.cells[r * d_c.columns + c] = oVal;
	//~~ALB
}

__global__ void logistic2D(Mat2D d_layer, Mat2D d_prev, int last = 0) {
	/*CUDA kernel for logistic nodes, this is the main forward (input to output) function

	  this function is going forward on the recursion path of the host function
	  needs: nodeset.h, math.h
	  */
	int r = blockDim.y * blockIdx.y + threadIdx.y; //getting row based on CUDA thread/block index
	int c = blockDim.x * blockIdx.x + threadIdx.x; //getting column based on CUDA thread/block index

	if (r < 1 && c < d_layer.columns) { //we are only updating the first layer (node outputs) of the array
		float l = d_layer.cells[d_layer.columns + c];//add in the bias

		/*we need to add up all of x*w of the previous layer, where:
				x is the output of a node in the previous layer, stored in row 1 of the previous layer
				w is the weight of the corresponding output, stored in column c of the current layer*/
		for (int i = 2; i < d_layer.rows;) { //start at 2 since 0 has the outputs and 1 has the bias

			l = l + d_layer.cells[i * d_layer.columns + c] * d_prev.cells[i - 2];
			
			printf("%i,%i>>l: %f, x: %f, w: %f\n", i -2, c, l, d_prev.cells[i - 2], d_layer.cells[i * d_layer.columns + c]);
			i++;
		}

		d_layer.cells[c] = 1 / (1 + exp(-l)); //now we calculate the output of this node
		printf("Row: %i, Col: %i, Input: %f, Value: %f \n", r, c, l, d_layer.cells[c]);

	}
	else
	{
		return;
	}
	//~~ALB
}

__global__ void nodeBackwardLog(Mat2D d_prev, Mat2D d_current, Mat2D d_next) {
	/*CUDA kernel for back propagation on logistic nodes
	the .dTh matrix is a matrix of the change in weights for the corresponding node-input pair
	the .dX matrix is a matrix of the change in the cascaded error term for the corresponding node-input pair
	the first row of .dX holds the sum  of the .dX terms in the next layer of the corresponding input

	This code is going backward (along the return path of the host recursive function)
	needs: nodeset.h, math.h
	*/
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;

	if (r > 0 && r < d_current.rows && c < d_current.columns) { //only operating on the weights and bias
		//printf("%f", d_current.dX[c]); //debug code
		
		float bX = d_current.cells[c]; //initialize variable for the dX/dL term
		bX = (bX * bX) - bX; //the (x^2 - x) term of dX/dL
		
		//printf("If 1 True\nR: %i, C: %i\n", r, c); //debug code
		//printf("d_next.columns %i\n", d_next.columns); //debug code

		float dbX_n = d_current.dX[c]; //the total dX/dx of the next layer is sumed into this layer in the first row of the .dX matrix
		//printf(" dbX_n %f ", dbX_n); //debug code

		if (r == 1) { //change to bias
			float dB = d_current.dTh[r * d_current.columns + c];
			dB = dB + bX * dbX_n;
			d_current.dTh[r * d_current.columns + c] = dB;
			d_current.dX[d_current.columns + c] = dbX_n;
		}
		else { //change to the weights (theta)
			float dTh = d_current.dTh[r * d_current.columns + c];
			float pX = d_prev.cells[r-2];
			dTh = dTh + dbX_n*bX*pX;
			d_current.dTh[r * d_current.columns + c] = dTh;
			d_current.dX[r*d_current.columns + c] = dbX_n * bX * d_current.cells[r* d_current.columns + c];

			d_prev.dX[r - 2] = d_prev.dX[r - 2] + d_current.dX[r * d_current.columns + c]; //sum the dX/dx layer for the previous layer
		}
	}
	else {
		return;
	}
	//~~ALB
}

__global__ void outPivotLog(Mat2D d_prev, Mat2D d_cur, Mat2D actual, int rn) {
	/*CUDA kernel for error comparison on logistic nodes
	the .dTh matrix is a matrix of the change in weights for the corresponding node-input pair
	the .dX matrix is a matrix of the change in the cascaded error term for the corresponding node-input pair
	the first row of .dX holds the sum  of the .dX terms in the next layer of the corresponding input

	This code is the pivot of the recursive function (aka from going forward to going backward)
	needs: nodeset.h, math.h
	*/
	
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;

	if (r > 0 && r < d_cur.rows && c < d_cur.columns) {
		//printf("If 1 True\nR: %i, C: %i\n", r, c); //debug code
		//printf("d_cur.columns %i\n", d_cur.columns); //debug code
		float bX = d_cur.cells[c];
		bX = (bX * bX) - bX;
		//printf("piv c: %i, run: %i, actual: %f\n", c, rn, actual.cells[rn * actual.columns + c]); //debug code
		float err = d_cur.cells[c] - actual.cells[rn * actual.columns + c];
		
		printf("Error Squared: %f\n", err * err);
		//printf("pre if 2\n");
		
		if (r == 1) { //change to bias
			//printf("True-2\n"); //debug code
			//printf("d_cur.columns %i", d_cur.columns); //debug code
			float dB = d_cur.dTh[r * d_cur.columns + c];
			dB = dB + bX * err;
			d_cur.dTh[r * d_cur.columns + c] = dB;
			d_cur.dX[d_cur.columns + c] = bX * err;
			//printf("dTh(%i, %i): %f, dX: %f\n", r, c, d_cur.dTh[r * d_cur.columns + c], d_cur.dX[d_cur.columns + c]); //debug code
		}
		else { //change to the weights (theta)
			//printf("Else-2\n"); //debug code
			float dTh = d_cur.dTh[r * d_cur.columns + c];
			float pX = d_prev.cells[c];
			dTh = dTh + err*bX*pX;
			d_cur.dTh[r * d_cur.columns + c] = dTh;
			d_cur.dX[r*d_cur.columns + c] = err * bX * d_cur.cells[r* d_cur.columns + c];

			d_prev.dX[r - 2] = d_prev.dX[r - 2] + d_cur.dX[r * d_cur.columns + c]; //sum the dX/dx layer for the previous layer
		}
	}
	else
	{
		return;
	}
	//~~ALB
}

__global__ void updateNodes(Mat2D d_nodes, float alpha) {
	/*CUDA kernel for weight/bias updates
	  This node is the update step of the learning process
	*/
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("Node Update: R- %i, C- %i, dnodes r- %i, dnodes c- %i \n", r, c, d_nodes.rows, d_nodes.columns);

	if (r > 0 && r < d_nodes.rows && c < d_nodes.columns) { //only update weights/biases
		float cur = d_nodes.cells[r * d_nodes.columns + c];
		float del = d_nodes.dTh[r * d_nodes.columns + c];
		d_nodes.cells[r * d_nodes.columns + c] = cur + (alpha * del);
		//printf(" %f ", d_nodes.cells[r * d_nodes.columns + c]); //debug code
	}
	else {
		return;
	}
	//~~ALB
}


Mat2D cudaMAdd2D(Mat2D f_mA, Mat2D f_mB) {
	if (f_mA.columns != f_mB.columns || f_mA.rows != f_mB.rows) {
		printf("ERROR: Incorrect array dimensions A + B. Sizes must be equal.\n");
		printf("A: %i x %i, B: %i x %i\n", f_mA.rows, f_mA.columns, f_mB.rows, f_mB.columns);
		Mat2D err;
		err.rows = -1;
		return err;
	}
	//Send input matricies to GPU and return d_mA so GPU memory can be deallocated later
	printf("--------- Addition ---------\n");
	Mat2D d_mA = cudaMSend2D(f_mA, true, "matrix A");
	Mat2D d_mB = cudaMSend2D(f_mB, true, "matrix B");

	//Create output matrix and allocate memory on GPU. Returns d_out to access result/deallocate mem
	Mat2D out;
	out.rows = d_mA.rows;
	out.columns = d_mA.columns;
	out.cells = (float*)malloc(out.rows * out.columns * sizeof(float));
	Mat2D d_out = cudaMSend2D(out, false, "Output matrix");

	//setup CUDA architecture and run kernel
	dim3 threadsPerBlock(16, 16); //each block will contain 16 by 16 threads
	dim3 numBlocks((d_out.columns + threadsPerBlock.x - 1) / threadsPerBlock.x, //number of blocks on x dimension of grid
		(d_out.rows + threadsPerBlock.y - 1) / threadsPerBlock.y); //number of blocks on y dimension of grid
	mAddKernel2D << <numBlocks, threadsPerBlock >> > (d_mA, d_mB, d_out); //run's kernal
	hipError_t errCode = hipDeviceSynchronize(); //synchronize cores to ensure everthing has been run
	printf("GPU Thread Synchronization: %s\n", hipGetErrorString(errCode));

	//debug code to find errors in execution of kernel
	errCode = hipGetLastError();
	if (errCode != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(errCode));
		exit(-1);
	}

	//retrieve output matrix from GPU memory
	errCode = hipMemcpy(out.cells, d_out.cells, d_out.rows * d_out.columns * sizeof(float), hipMemcpyDeviceToHost);
	printf("Pulling Output matrix from GPU: %s\n", hipGetErrorString(errCode));

	//print result
	print2DMat(out, "\n\n--Addition Results--\nOutput ");
	printf("--------------------------------\n");

	//deallocate GPU memory
	hipFree(d_mA.cells);
	hipFree(d_mB.cells);
	hipFree(d_out.cells);

	return out;
	//~~ALB
}

Mat2D cudaMSub2D(Mat2D f_mA, Mat2D f_mB) {
	if (f_mA.columns != f_mB.columns || f_mA.rows != f_mB.rows) {
		printf("ERROR: Incorrect array dimensions A + B. Sizes must be equal.\n");
		printf("A: %i x %i, B: %i x %i\n", f_mA.rows, f_mA.columns, f_mB.rows, f_mB.columns);
		Mat2D err;
		err.rows = -1;
		return err;
	}
	//Send input matricies to GPU and return d_mA so GPU memory can be deallocated later
	printf("--------- Subtraction ---------\n");
	Mat2D d_mA = cudaMSend2D(f_mA, true, "matrix A");
	Mat2D d_mB = cudaMSend2D(f_mB, true, "matrix B");

	//Create output matrix and allocate memory on GPU. Returns d_out to access result/deallocate mem
	Mat2D out;
	out.rows = d_mA.rows;
	out.columns = d_mA.columns;
	out.cells = (float*)malloc(out.rows * out.columns * sizeof(float));
	Mat2D d_out = cudaMSend2D(out, false, "Output matrix");

	//setup CUDA architecture and run kernel
	dim3 threadsPerBlock(16, 16); //each block will contain 16 by 16 threads
	dim3 numBlocks((d_out.columns + threadsPerBlock.x - 1) / threadsPerBlock.x, //number of blocks on x dimension of grid
		(d_out.rows + threadsPerBlock.y - 1) / threadsPerBlock.y); //number of blocks on y dimension of grid
	mSubKernel2D <<<numBlocks, threadsPerBlock >>> (d_mA, d_mB, d_out); //run's kernal
	hipError_t errCode = hipDeviceSynchronize(); //synchronize cores to ensure everthing has been run
	printf("GPU Thread Synchronization: %s\n", hipGetErrorString(errCode));

	//debug code to find errors in execution of kernel
	errCode = hipGetLastError();
	if (errCode != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(errCode));
		exit(-1);
	}

	//retrieve output matrix from GPU memory
	errCode = hipMemcpy(out.cells, d_out.cells, d_out.rows * d_out.columns * sizeof(float), hipMemcpyDeviceToHost);
	printf("Pulling Output matrix from GPU: %s\n", hipGetErrorString(errCode));

	//print result
	print2DMat(out, "\n\n--Subtraction Results--\nOutput ");
	printf("--------------------------------\n");

	//deallocate GPU memory
	hipFree(d_mA.cells);
	hipFree(d_mB.cells);
	hipFree(d_out.cells);

	return out;
	//~~ALB
}

Mat2D cudaMTrans2D(Mat2D f_m) {
	printf("--------- Transposition ---------\n");
	Mat2D d_m = cudaMSend2D(f_m, true, "Original");

	Mat2D out;
	//out.id = getID(mIds, 1);
	out.rows = f_m.columns;
	out.columns = f_m.rows;
	out.cells = (float*)malloc(out.rows*out.columns * sizeof(float));
	Mat2D d_o = cudaMSend2D(out, true, "output");

	dim3 tPBlock(16, 16);
	dim3 nBlocks((out.columns + tPBlock.x - 1) / tPBlock.x, (out.rows + tPBlock.y - 1) / tPBlock.y);
	mTransKernel2D << <nBlocks, tPBlock >> > (d_m, d_o);
	hipError_t errCode = hipDeviceSynchronize(); //synchronize cores to ensure everthing has been run
	printf("GPU Thread Synchronization: %s\n", hipGetErrorString(errCode));

	//debug code to find errors in execution of kernel
	errCode = hipGetLastError();
	if (errCode != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(errCode));
		exit(-1);
	}

	//retrieve output matrix from GPU memory
	errCode = hipMemcpy(out.cells, d_o.cells, d_o.rows * d_o.columns * sizeof(float), hipMemcpyDeviceToHost);
	printf("Pulling Output matrix from GPU: %s\n", hipGetErrorString(errCode));

	//print result
	print2DMat(out, "\n\n--Transposition Results--\nOutput ");
	printf("--------------------------------\n");

	hipFree(d_m.cells);
	hipFree(d_o.cells);

	return out;


	//~~ALB
}

Mat2D cudaMMult2D(Mat2D f_mA, Mat2D f_mB) { 
	/*2D Matrix multiplication algorithm
	  needs: mMultKernel2D<<<nBlocks, tPb>>>(Mat2D d_mA, Mat2D d_mB, Mat2D d_out),
	  cudaMSend2D(Mat2D f_mX, Bool TF, const char* ID), nodeSet.h

	  Adapted from:
	  -Robert Hochberg (1/24/16): http://bit.ly/2iA8jDc
	  */

	//check dimensions, err if incorrect
	if (f_mA.columns != f_mB.rows) {
		printf("ERROR: Incorrect array dimensions A*B. Number of columns in A must equal number of rows in B.\n");
		printf("A: %i x %i, B: %i x %i\n", f_mA.rows, f_mA.columns, f_mB.rows, f_mB.columns);
		Mat2D err;
		err.rows = -1;
		return err;
	}

	//Send input matricies to GPU and return d_mA so GPU memory can be deallocated later
	printf("---------Multiplication---------\n");
	Mat2D d_mA = cudaMSend2D(f_mA, true, "matrix A");
	Mat2D d_mB = cudaMSend2D(f_mB, true, "matrix B");

	//Create output matrix and allocate memory on GPU. Returns d_out to access result/deallocate mem
	Mat2D out;
	//out.id = getID(master, 1);
	out.rows = d_mA.rows;
	out.columns = d_mB.columns;
	out.cells = (float*)malloc(out.rows * out.columns * sizeof(float));
	Mat2D d_out = cudaMSend2D(out, false, "Output matrix");

	//setup CUDA architecture and run kernel
	dim3 threadsPerBlock(16, 16); //each block will contain 16 by 16 threads
	dim3 numBlocks((d_out.columns + threadsPerBlock.x - 1) / threadsPerBlock.x, //number of blocks on x dimension of grid
		(d_out.rows + threadsPerBlock.y) / threadsPerBlock.y); //number of blocks on y dimension of grid
	mMultKernel2D << <numBlocks, threadsPerBlock >> > (d_mA, d_mB, d_out); //run's kernal
	hipError_t errCode = hipDeviceSynchronize(); //synchronize cores to ensure everthing has been run
	printf("GPU Thread Synchronization: %s\n", hipGetErrorString(errCode));

	//debug code to find errors in execution of kernel
	errCode = hipGetLastError();
	if (errCode != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(errCode));
		exit(-1);
	}

	//retrieve output matrix from GPU memory
	errCode = hipMemcpy(out.cells, d_out.cells, d_out.rows * d_out.columns * sizeof(float), hipMemcpyDeviceToHost);
	printf("Pulling Output matrix from GPU: %s\n", hipGetErrorString(errCode));

	//print result
	print2DMat(out, "\n\n--Multiplication Results--\nOutput ");
	printf("--------------------------------\n");

	//deallocate GPU memory
	hipFree(d_mA.cells);
	hipFree(d_mB.cells);
	hipFree(d_out.cells);

	return out;
	//~~ALB
}

Mat2D layerSetup(laySet setup, int indx, bool onesFirst = true) { 
	/*Setup of the node layers:
	  Each array has the output node in row zero, and the weights for each input in the rows below.
	  So, # of rows = # of nodes from previous layer + 2 (the output node row and a bias row on row 1)
	  This also means, # of rows = # of columns of previous layer + 2
	*/
	Mat2D out;
	out.columns = setup.nPl[indx]; 
	out.rows = 1;
	out.rrows = 1;
	/* The first layer will have no weight or bias, and therefore will have 
	   only one row	*/
	if (indx == setup.layers - 1) {
		out.rows = setup.nPl[indx - 1] + 2;
	}
	else if (indx != 0)
	{	
		out.rows = setup.nPl[indx - 1] + 2;
	}

	//allocate memory for cells
	out.cells = (float*)malloc(out.rows * out.columns * sizeof(float));
	out.dTh = (float*)malloc(out.rows * out.columns * sizeof(float));
	out.dX = (float*)malloc(out.rows * out.columns * sizeof(float));
	//initialize array with 1's in row 0 and zeros in rest
	float zer = 1;
	for (int r = 0; r < out.rows; ++r) {
		for (int c = 0; c < out.columns; ++c) {
			out.cells[r * out.columns + c] = zer;
			out.dTh[r * out.columns + c] = 0;
			out.dX[r * out.columns + c] = 0;
		}
		zer = 0;
	}

	return out;
	//~~ALB
}

Mat2D* hiddenSetup(laySet setup) {
	/*Setup of the 'hidden' layers and input layer
	  The goal is to setup a linked list. Its the dream..
	  From here on we're passing pointers rather than the structure
	*/
	printf("\n\n============= Hidden Layers Setup ===============\n");
	Mat2D *first;
	Mat2D *prev;
	Mat2D *next;
	
	/*The below code is my way of setting up a linked list. 
	  I think there's a better way to code this without dereferencing so much,
	  but it works, so for now I'm not changing it :) */
	int i = 0;

	first = (Mat2D*)malloc(sizeof(Mat2D));//allocate memory
	*first = layerSetup(setup, i);//setup first layer, we're going to hold on to the first layer
	printf("layer %i\n", i);
	print2DMat(*first);
	printf("layer %i-- rows:%i, cols:%i\n\n", i, first->rows, first->columns);
	i++;

	prev = (Mat2D*)malloc(sizeof(Mat2D));//allocate memory
	*prev = layerSetup(setup, i); //setup second layer
	printf("layer %i\n", i);
	print2DMat(*prev);
	printf("layer %i-- rows:%i, cols:%i\n\n", i, prev->rows, prev->columns);
	i++;

	(*first).next = (struct Mat2D*)prev; //Link first to 2nd
	
	/*Iterate through the layer setup and make the layers*/
	for (; i < setup.layers;) {
		next = (Mat2D*)malloc(sizeof(Mat2D)); //allocate memory
		*next = layerSetup(setup, i); //setup ith layer
		printf("layer %i\n", i);
		print2DMat(*next);
		printf("layer %i-- rows:%i, cols:%i\n\n", i, next->rows, next->columns);
		(*prev).next = (struct Mat2D*)next; //link i-1'th to ith
		prev = next;
		i++;
	}
	(*next).next = NULL;//give the last layer's next a null pointer
	first->end = next;
	//return the first layer as it will have the links  to all
	return first;

	//~~ALB
}

Mat2D* lmatSend2D(Mat2D* nodes) {
	/*Code for sending the linked matricies to the GPU.
	  We cant use the CudaMsend2D since we want to use pointers.
	  This needs to eventually be moved to the header I think, but for now it stays.
	  */

	Mat2D* d_nodes = (Mat2D*)malloc(sizeof(Mat2D));//This time we have to allocate memory for the device pointer

	hipError_t errCode = hipMalloc(&d_nodes->cells, nodes->rows * nodes->columns * sizeof(float)); //allocate mem on GPU
	printf("GPU hipMalloc Nodes: %s\n", hipGetErrorString(errCode));
	errCode = hipMalloc(&d_nodes->dTh, nodes->rows * nodes->columns * sizeof(float)); //allocate mem on GPU
	printf("GPU hipMalloc dTh: %s\n", hipGetErrorString(errCode));
	errCode = hipMalloc(&d_nodes->dX, nodes->rows * nodes->columns * sizeof(float)); //allocate mem on GPU
	printf("GPU hipMalloc dX: %s\n", hipGetErrorString(errCode));
	d_nodes->rows = nodes->rows;
	d_nodes->columns = nodes->columns;
	
	errCode = hipMemcpy(d_nodes->cells, nodes->cells, d_nodes->rows * d_nodes->columns * sizeof(float), hipMemcpyHostToDevice);//copy cell values to GPU
	printf("Memcpy Nodes: %s\n", hipGetErrorString(errCode));
	errCode = hipMemcpy(d_nodes->dTh, nodes->dTh, d_nodes->rows * d_nodes->columns * sizeof(float), hipMemcpyHostToDevice);//copy cell values to GPU
	printf("Memcpy dTh: %s\n", hipGetErrorString(errCode));
	errCode = hipMemcpy(d_nodes->dX, nodes->dX, d_nodes->rows * d_nodes->columns * sizeof(float), hipMemcpyHostToDevice);//copy cell values to GPU
	printf("Memcpy dX: %s\n", hipGetErrorString(errCode));
	return d_nodes;
	//~~ALB
}

Mat2D* sendActual(Mat2D* actual) {
	/*A function to send the training set output values to the GPU for comparison
	
	*/
	Mat2D* d_act = (Mat2D*)malloc(sizeof(Mat2D));
	printf("2: %f, 4: %f\n", actual->cells[2], actual->cells[4]);
	hipError_t errCode = hipMalloc(&d_act->cells, actual->rows *  actual->columns * sizeof(float));
	printf("GPU hipMalloc Actual Values: %s\n", hipGetErrorString(errCode));
	d_act->rows = actual->rows;
	d_act->columns = actual->columns;
	errCode = hipMemcpy(d_act->cells, actual->cells, actual->rows * actual->columns * sizeof(float), hipMemcpyHostToDevice);
	printf("Memcpy Actual Nodes: %s\n", hipGetErrorString(errCode));
	return d_act;
	//~~ALB
}

Mat2D* nodeRetrieve(Mat2D* d_nodes, Mat2D* nodes, bool free = true) {
	/*Code for retrieving layer arrays from GPU
	*/
	Mat2D* first = nodes;// this will be the output

	//Get values from GPU
	hipError_t errCode = hipMemcpy(first->cells, d_nodes->cells, d_nodes->rows * d_nodes->columns * sizeof(float), hipMemcpyDeviceToHost);
	printf("Retrieving nodes from GPU: %s\n", hipGetErrorString(errCode));
	errCode = hipMemcpy(first->dTh, d_nodes->dTh, d_nodes->rows * d_nodes->columns * sizeof(float), hipMemcpyDeviceToHost);
	printf("Retrieving updates from GPU: %s\n", hipGetErrorString(errCode));

	//Debug code (see very bottom of this file for more helpful debug code)
	errCode = hipGetLastError();
	if (errCode != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(errCode));
		exit(-1);
	}


	Mat2D* d_temp = d_nodes; // for freeing if applicable
	d_nodes = d_nodes->next;//move through linked list

	if (free) {
		hipFree(d_temp->cells);//free gpu memory if applicable
		hipFree(d_temp->dTh);
		hipFree(d_temp->dX);
	}
	nodes = nodes->next;//move through nodes

	while (nodes != NULL) {
		//get values from GPU... everything in this loop is basically the same as above
		errCode = hipMemcpy(nodes->cells, d_nodes->cells, d_nodes->rows * d_nodes->columns * sizeof(float), hipMemcpyDeviceToHost);
		printf("Retrieving nodes from GPU: %s\n", hipGetErrorString(errCode));
		errCode = hipMemcpy(nodes->dTh, d_nodes->dTh, d_nodes->rows * d_nodes->columns * sizeof(float), hipMemcpyDeviceToHost);
		printf("Retrieving updates from GPU: %s\n", hipGetErrorString(errCode));
		d_temp = d_nodes;
		d_nodes = d_nodes->next;
		if (free) {
			hipFree(d_temp->cells);//free gpu memory if applicable
			hipFree(d_temp->dTh);
			hipFree(d_temp->dX);
		}
		nodes = nodes->next;
	}
	return first;
	//~~ALB
}

Mat2D* pNodesSetup(Mat2D* nodes) {
	/*Code for processing the nodes.
	  This should eventually be improved with aSync.
	  The idea here is to be able to try to keep the transfers to the GPU to a minimum
	  Eventually some of this code should be put onto the GPU
	  */
	printf("\n\n============= Process Nodes Setup ===============\n");
	Mat2D* first = nodes;
	Mat2D* next = first->next;

	Mat2D* d_first = lmatSend2D(first); //send first, and get back pointer to device-first
	Mat2D* d_next = lmatSend2D(next); //send next, and get back pointer to device-next
	d_first->next = d_next; // yep, we're creating a linked list to retrieve everything later

	dim3 threadsPerBlock(16, 16); //standard tpb code
	dim3 numBlocks((d_next->rows + threadsPerBlock.y)/threadsPerBlock.y,
					(d_next->columns + threadsPerBlock.x)/threadsPerBlock.x); //standard nblocks code

	printf("Layer 1\n"); //I didn't print a layer 0 because nothing is going on there
	logistic2D <<<numBlocks, threadsPerBlock >>> (*d_next, *d_first); // kernal execution
	hipError_t errCode = hipDeviceSynchronize(); //sync threads
	printf("GPU Thread Synchronization: %s\n", hipGetErrorString(errCode));


	Mat2D* d_prev = d_next;
	next = next->next;//move through linked list

	int i = 2;//for tracking in the print code
	while (next != NULL) { //go until end of host linked list
		d_next = lmatSend2D(next);//send next layer
		d_prev->next = d_next;//building device linked list

		//basically repeated from above
		dim3 numBlocks((d_next->rows + threadsPerBlock.y) / threadsPerBlock.y,
					   (d_next->columns + threadsPerBlock.x) / threadsPerBlock.x);
		printf("Layer %i\n", i);
		logistic2D <<<numBlocks, threadsPerBlock >>> (*d_next, *d_prev);
		errCode = hipDeviceSynchronize();
		printf("GPU Thread Synchronization: %s\n", hipGetErrorString(errCode));

		//next
		d_prev = d_next;
		next = next->next; //move through linked list
		i++;
	}
	d_prev->next = NULL;
	return d_first; //return device linked list so we can get the results later
					//~~ALB
}

Mat2D* uNodes(Mat2D* d_nodes, float alpha) {
	/*Code to update the node weights during the learning cycle
	*/

	Mat2D* t = d_nodes;
	printf("==================STARTING NODE UPDATE========================= \n");
	int i = 0;
	//go through linked list of layer arrays
	while (t != NULL) {
		dim3 tPb(BLKSZ, BLKSZ);
		dim3 nb((t->rows + tPb.y) / tPb.y, (t->columns + tPb.x) / tPb.x);
		
		printf("\nLayer %i Update \n", i);
		updateNodes<<<nb, tPb>>>(*t, alpha);
		
		hipError_t errCode = hipDeviceSynchronize();
		printf("\nNode Update: %s\n", hipGetErrorString(errCode));
		
		errCode = hipGetLastError();
		if (errCode != hipSuccess)
		{
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(errCode));
			exit(-1);
		}
		t = t->next;
		i++;
	}
	hipError_t errCode = hipDeviceSynchronize();
	printf("\nNode Update: %s\n", hipGetErrorString(errCode));
	printf("==================NODE UPDATE COMPLETE========================= \n");
	return d_nodes;
	//~~ALB
}

Mat2D* changeIn(Mat2D* d_nodes, Mat2D* inputs, int r) {
	//function to change the input layer for each cycle through
	printf("-Change Inputs-\n");
	Mat2D* d_in = (Mat2D*)malloc(sizeof(Mat2D));
	d_in->cells = (float*)malloc(sizeof(inputs->cells));

	int i = 0;
	for (; i < inputs->columns;) {
		d_in->cells[i] = inputs->cells[r * inputs->columns + i];
		r++;
		i++;
	}
	hipFree(d_nodes->cells);
	hipError_t errCode = hipMalloc(&d_nodes->cells, sizeof(inputs));
	printf("Input change malloc: %s\n", hipGetErrorString(errCode));
	errCode = hipMemcpy(d_nodes->cells, d_in->cells, sizeof(inputs), hipMemcpyHostToDevice);
	printf("Input change memcpy: %s\n\n", hipGetErrorString(errCode));
	return d_nodes;
	//~~ALB
}

Mat2D* pNodes(Mat2D* d_n, bool learn = false, Mat2D* actual = NULL, int run = 0, Mat2D* last = NULL) {
	/* Main processing function... 

	Travels through the linked list, calling the main logistic forward function
	Reaches the end and pivots, comparing the output with the actual value in the training set
	Travels backwards through the recursion return path and calculates the change to the node weights
	
	*/

	//Travels through the linked list, calling the main logistic forward function
	if (d_n->next != NULL) {
		printf("-Calc Forward-\n");
		dim3 tPb(BLKSZ, BLKSZ);
		dim3 nb((d_n->rows + tPb.y) / tPb.y, (d_n->columns + tPb.x) / tPb.x);
		logistic2D <<< nb, tPb >>> (*d_n->next, *d_n);
		hipError_t errCode = hipDeviceSynchronize();
		printf("Calc Forward STATUS: %s\n\n", hipGetErrorString(errCode));
		d_n = pNodes(d_n->next, learn, actual, run, d_n);
	}

	//Reaches the end and pivots, comparing the output with the actual value in the training set
	else if (learn) {
		//printf("-Calc Pivot-\n");
		dim3 tPb(BLKSZ, BLKSZ);
		dim3 nb((d_n->rows + tPb.y) / tPb.y, (d_n->columns + tPb.x) / tPb.x);
		outPivotLog <<< nb, tPb >>> (*last, *d_n, *actual, run);
		hipError_t errCode = hipDeviceSynchronize();
		printf("Calc Pivot STATUS: %s\n\n", hipGetErrorString(errCode));
	}
	

	//Travels backwards through the recursion return path and calculates the change to the node weights
	if (learn && d_n->next != NULL && last != NULL) {
		//printf("-Calc Backwards-\n");
		//printf("dX:\n");
		dim3 tPb(BLKSZ, BLKSZ);
		dim3 nb((d_n->rows + tPb.y) / tPb.y, (d_n->columns + tPb.x) / tPb.x);
		nodeBackwardLog <<< nb, tPb >>> (*last, *d_n, *d_n->next);
		hipError_t errCode = hipDeviceSynchronize();
		printf("\n Calc Backwards STATUS: %s\n\n", hipGetErrorString(errCode));
	}

	if (last != NULL) {
		return last;
	}
	else {
		return d_n;
	}
	//~~ALB
}


int main(int argc, char* argv) {
	//These testers for the matrix math will eventually be gone
	//float a[] = {
	//	2, 4, 6, 7,
	//	1, 3, 4, 6, 9
	//};
	//int rA = 3;
	//int cA = 3;
	//float b[] = {
	//	0, 4,
	//    2, 3, 
	//	2, 4,
	//	4, 4, 3
	//};
	//int rB = 3;
	//int cB = 3;

	printf("============= Initializing ===============\n");
	//Below is a layer setup variable, this takes an array where each value is the number of nodes in that layer
	laySet lay;
	int x[] = { 1, 5, 3, 1};
	lay.layers = sizeof(x) / sizeof(int);
	lay.nPl = x;

	//below is the training set inputs as a vector (turned into a Mat2D)... this is currently just hard-coded.
	//each row is the next input for the traning set
	Mat2D* inputs = (Mat2D*)malloc(sizeof(Mat2D));
	inputs->columns = x[0];
	float inp[] = { 1, 5, 3, 2, 3, 4, 5, 1, 1, 5, 3, 2, 3, 4, 5, 1, 1, 5, 3, 2, 3, 4, 5, 1 };
	inputs->cells = (float*)malloc(sizeof(inp));
	inputs->cells = inp;

	//below is the training actual values set as a vector... this is currently just hard-coded.
	//each row is the next actual value for the traning set
	Mat2D* actual = (Mat2D*)malloc(sizeof(Mat2D));
	actual->columns = lay.nPl[lay.layers -1];
	float act[] = { 0.5, 2.5, 1.5, 1, 1.5, 2, 2.5, 0.5, 0.5, 2.5, 1.5, 1, 1.5, 2, 2.5, 0.5, 0.5, 2.5, 1.5, 1, 1.5, 2, 2.5, 0.5 };
	actual->rows = sizeof(act) / sizeof(float);
	vecToMat2DP(act, actual);

	//actual->cells = (float*)malloc(sizeof(act));
	//actual->cells = act;
	//print2DMat(*actual);
	

	int bSize = 4; //batch size.... the number of runs before updating the variables

	Mat2D* first = hiddenSetup(lay); //setup the hidden layers based on the layset variable lay
	Mat2D* d_first = pNodesSetup(first); //initial run through with all inputs set to one
	Mat2D* d_act = sendActual(actual); //send the training set outputs to the GPU


	//now we run through all of the training set... This will later be replaced by other options as to when 
	//the learning stops
	int rn = 0;
	printf("\n\n========================================================== Begin Batch Run =====================================================\n");
	printf("---Batch Size: %i\n", bSize);
	for (; rn < sizeof(inp)/sizeof(float);) {
		int b = bSize;
		while (b > 0) { //this loops until the batch size is met and then moves to the update step
			printf("\n--Run: %i\n", rn);
			d_first = changeIn(d_first, inputs, rn); //change the inputs to the corresponding training set
			d_first = pNodes(d_first, true, d_act, rn); //process the nodes
			b = b - 1; //next batch index
			rn = rn + 1; //next run index
		}
		d_first = uNodes(d_first, -0.5/bSize); //update the node weights
	}
	printf("\n============================================================= End Batch Run ======================================================\n");
	hipError_t errCode = hipGetLastError();
	printf("Starting nodeRetrieve, last CUDA error: %s\n", hipGetErrorString(errCode));

	first = nodeRetrieve(d_first, first);
	
	Mat2D* temp;
	int i = 0;
	while (first != NULL) {
		printf("Layer %i\n", i);
		print2DMat(*first);
		temp = (Mat2D*)first;
		first = first->next;
		free(temp);
		i++;
		
	}

//++++++++!!!!!!!! If having errors on external code, the reset below may cause it================
	errCode = hipDeviceReset(); //clear any remaining items on device...
	printf("GPU reset: %s\n", hipGetErrorString(errCode));
	return 0;
	}


/*Austin's useful debugging tools
=======================================================================
This one is good just to put in the code for status:

hipError_t errCode = ...
printf("Retrieving nodes from GPU: %s\n", hipGetErrorString(errCode));

========================================================================
This one is good to put after kernal execution to get any errors w/in the kernal:
Put it after a thread sync call

hipError_t errCode = hipGetLastError();
if (errCode != hipSuccess)
{
fprintf(stderr, "ERROR: %s\n", hipGetErrorString(errCode));
exit(-1);
}


========================================================================
Below is some good console debug code for errors within the kernal:

nvcc -lineinfo -o matops matops.cu
cuda-memcheck ./matops |more

========================================================================
Below is a good way of deciphering error messages:
www.google.com



~~ALB
*/